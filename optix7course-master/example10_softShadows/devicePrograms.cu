#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "myString.cuh"

#include "LaunchParams.h"
#include "gdt/random/random.h"
#include "TriangleData.cuh"

//#define PARALLEL_LIGHT

using namespace osc;

#define NUM_LIGHT_SAMPLES 1
#define NUM_PIXEL_SAMPLES 8
constexpr int RRBeginDepth = 4;
#define maxDepth 7

__device__ vec3f missColor;

namespace osc
{

    typedef gdt::LCG<16> Random;

    /*! launch parameters in constant memory, filled in by optix upon
        optixLaunch (this gets filled in from the buffer we pass to
        optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;

    /*! per-ray data now captures random number generator, so programs
        can access RNG state */
    struct PRD
    {
        Random random;
        vec3f pixelColor;
        vec3f nextRayOrigin;
        vec3f nextRayDirection;
        vec3i ch_triangle_index; //only updated when hit glass for now
        int depth;
        char pathREGEX[maxDepth] = {};

        bool isEnd;
        bool anyHitLight;
    };

    static __forceinline__ __device__
    void *unpackPointer(uint32_t i0, uint32_t i1)
    {
        const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
        void *ptr = reinterpret_cast<void *>( uptr );
        return ptr;
    }

    static __forceinline__ __device__
    void packPointer(void *ptr, uint32_t &i0, uint32_t &i1)
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    static __forceinline__ __device__ T *getPRD()
    {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T *>( unpackPointer(u0, u1));
    }

    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    extern "C" __global__ void __closesthit__shadow()
    {
        /* not going to be used ... */
    }

    struct Onb
    {
        __forceinline__ __device__ Onb(const vec3f &normal)
        {
            m_tangent = vec3f(0.f);
            m_binormal = vec3f(0.f);
            m_normal = vec3f(0.f);

            m_normal = normal;

            if (fabs(m_normal.x) > fabs(m_normal.z))
            {
                m_binormal.x = -m_normal.y;
                m_binormal.y = m_normal.x;
                m_binormal.z = 0;
            }
            else
            {
                m_binormal.x = 0;
                m_binormal.y = -m_normal.z;
                m_binormal.z = m_normal.y;
            }

            m_binormal = normalize(m_binormal);
            m_tangent = cross(m_binormal, m_normal);
        }

        __forceinline__ __device__ void inverse_transform(vec3f &p) const
        {
            p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
        }

        vec3f m_tangent;
        vec3f m_binormal;
        vec3f m_normal;
    };

static __device__ __inline__ vec3f reflect(vec3f rayDir, vec3f normal)
{
    return rayDir - 2.f * dot(normal, rayDir) * normal;
}

static __device__ __inline__ vec3f refract(vec3f rayDir, vec3f normal, float eta)
{
    const float k = 1.f - eta * eta * (1.f - dot(normal, rayDir) * dot(normal, rayDir));
    if(k < 0.f)
        return vec3f(0.f);
    else
        return eta * rayDir - (eta * dot(normal, rayDir) + sqrt(k)) * normal;
}

static __device__ __inline__ bool refract(vec3f &w_t, vec3f rayDir, vec3f normal, float eta)
{
    const float k = 1.f - eta * eta * (1.f - dot(normal, rayDir) * dot(normal, rayDir));
    if(k < 0.f)
    {
        w_t = vec3f(0.f);
        return false;
    }
    else
    {
        w_t = eta * rayDir - (eta * dot(normal, rayDir) + sqrt(k)) * normal;
        return true;
    }
}

static __device__ __inline__ float fresnel(float cos_theta_i, float cos_theta_t, float eta)
{
    const float rs = (cos_theta_i - cos_theta_t * eta) / (cos_theta_i + eta * cos_theta_t);
    const float rp = (cos_theta_i * eta - cos_theta_t) / (cos_theta_i * eta + cos_theta_t);

    return 0.5f * (rs * rs + rp * rp);
}

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, vec3f& p)
{
    // Uniformly sample disk.
    const float r   = sqrtf( u1 );
    const float phi = 2.0f * 3.141592653589793 * u2;
    p.x = r * cosf( phi );
    p.y = r * sinf( phi );

    // Project up to hemisphere.
    p.z = sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) );
}


extern "C" __global__ void __anyhit__radiance() { /*! for this simple example, this will remain empty */ }

extern "C" __global__ void __anyhit__shadow() { /*! not going to be used */ }

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance()
{
    PRD &prd = *getPRD<PRD>();
    // set to constant white as background color
    prd.isEnd = true;
    prd.pixelColor *= missColor;
}

extern "C" __global__ void __miss__shadow()
{
    // we didn't hit anything, so the light is visible
    vec3f &prd = *(vec3f *) getPRD<vec3f>();
    prd = vec3f(0.f);
}

__device__ bool init = false;
//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x; //frame buffer index

    PPD &nowPPD = optixLaunchParams.frame.ppdBuffer[fbIndex]; //get the per pixel data of this pixel


    const int accumID = optixLaunchParams.frame.accumID;
    const auto &camera = optixLaunchParams.camera;

    if(accumID == 0 && !init)
    {
        init = true;
        missColor = vec3f(0.f);
    }

    PRD prd;
    prd.random.init(ix + accumID * optixLaunchParams.frame.size.x,
                    iy + accumID * optixLaunchParams.frame.size.y);

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    int numPixelSamples = NUM_PIXEL_SAMPLES;

    vec3f pixelColor = 0.f;
    for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
    {
        //clear prd
        prd.isEnd = false;
        prd.depth = 0;
        prd.nextRayDirection = vec3f(0.f);
        prd.nextRayOrigin = vec3f(0.f);
        my_strcpy(prd.pathREGEX, "");

        // normalized screen plane position, in [0,1]^2
        const vec2f screen(vec2f(ix + prd.random(), iy + prd.random())
                           / vec2f(optixLaunchParams.frame.size));

        // generate ray direction
        vec3f rayDir = normalize(camera.direction
                                 + (screen.x - 0.5f) * camera.horizontal
                                 + (screen.y - 0.5f) * camera.vertical);
        vec3f rayOrigin = camera.position;
        prd.pixelColor = vec3f(1.f);

        for (int i = 0; !prd.isEnd; i++)
        {
            optixTrace(optixLaunchParams.traversable,
                       rayOrigin,
                       rayDir,
                       0.0001f,    // tmin
                       1e20f,  // tmax
                       0.0f,   // rayTime
                       OptixVisibilityMask(255),
                       OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                       RADIANCE_RAY_TYPE,            // SBT offset
                       RAY_TYPE_COUNT,               // SBT stride
                       RADIANCE_RAY_TYPE,            // missSBTIndex
                       u0, u1);

            if(prd.depth >= RRBeginDepth)
            {
                float p = length(prd.pixelColor);
                p = min(p , 1.f);
                if(prd.random() >= p)
                    break;
                prd.pixelColor /= p;
            }
            if(prd.depth >= maxDepth)
                break;

            if(prd.isEnd)
                break;

            rayOrigin = prd.nextRayOrigin;
            rayDir = prd.nextRayDirection;
        }
        if(prd.isEnd)
        {
            pixelColor += prd.pixelColor;
        }
    }
    // and write to frame buffer ...
    if (accumID == 0)
    {
        const int r = int(255.99f * min(pixelColor.x / numPixelSamples, 1.f));
        const int g = int(255.99f * min(pixelColor.y / numPixelSamples, 1.f));
        const int b = int(255.99f * min(pixelColor.z / numPixelSamples, 1.f));

        // convert to 32-bit rgba value (we explicitly set alpha to 0xff
        // to make stb_image_write happy ...
        const uint32_t rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);
        optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
        optixLaunchParams.frame.accumulateBuffer[fbIndex] = pixelColor / numPixelSamples;
    }
    else
    {
        vec3f prevColor = optixLaunchParams.frame.accumulateBuffer[fbIndex];
        vec3f newColor = prevColor + (((pixelColor / numPixelSamples) - prevColor) / (accumID + 1));

        optixLaunchParams.frame.accumulateBuffer[fbIndex] = newColor;
        const int r = int(255.99f * min(newColor.x, 1.f));
        const int g = int(255.99f * min(newColor.y, 1.f));
        const int b = int(255.99f * min(newColor.z, 1.f));

        // convert to 32-bit rgba value (we explicitly set alpha to 0xff
        // to make stb_image_write happy ...
        const uint32_t rgba = 0xff000000 | (r << 0) | (g << 8) | (b << 16);
        optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
    }
}

extern "C" __global__ void __closesthit__radiance()  //diffuse
{
    const TriangleMeshSBTData &sbtData
            = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();
    prd.depth++;

    const TriangleData triangleData(sbtData);

    // start with some ambient term
    vec3f pixelColor = triangleData.diffuseColor;

    const float z1 = prd.random();
    const float z2 = prd.random();
    vec3f w_in = normalize(vec3f(1, 1, 1));
    cosine_sample_hemisphere( z1, z2, w_in );
    Onb onb(triangleData.Ns);
    onb.inverse_transform( w_in );

    prd.nextRayOrigin = triangleData.surfPos + 1e-3f * triangleData.Ns;
    prd.nextRayDirection = w_in;
    prd.pixelColor *= pixelColor;
    my_strcat(prd.pathREGEX, "D");
}

extern "C" __global__ void __closesthit__metal()
{
    const TriangleMeshSBTData &sbtData
            = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();

    const TriangleData triangleData(sbtData);

    vec3f reflectDirection = reflect(triangleData.rayDir, triangleData.Ns);

    prd.nextRayOrigin = triangleData.surfPos + 1e-3f * triangleData.Ns;
    prd.nextRayDirection = reflectDirection;
    prd.depth++;
    my_strcat(prd.pathREGEX, "M");
}

extern "C" __global__ void __closesthit__glass()
{
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x; //frame buffer index

    PPD &nowPPD = optixLaunchParams.frame.ppdBuffer[fbIndex]; //get the per pixel data of this pixel

    const TriangleMeshSBTData &sbtData
            = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();

    const TriangleData triangleData(sbtData);


    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    vec3f frontFacedNormal = triangleData.rawNormal;
    float wavelengthIor = sbtData.refractionIndex;

    float cos_theta_i = dot(-triangleData.rayDir, frontFacedNormal);
    float eta;
    float t_hit = optixGetRayTmax();

    vec3f extinction(-1 * log(0.9f), -1 * log(0.9f), -1 * log(0.9f));
    vec3f transmittance(1.f);
    if(cos_theta_i > 0.f)
    {
        // Ray is entering
        eta = wavelengthIor;// Note: does not handle nested dielectrics
    }
    else
    {
        // Ray is exiting; apply Beer's Law.
        // This is derived in Shirley's Fundamentals of Graphics book.
        transmittance = vec3f(expf(-extinction.x * t_hit), expf(-extinction.y * t_hit), expf(-extinction.z * t_hit));

        eta = 1.f / wavelengthIor;
        cos_theta_i = -cos_theta_i;
        frontFacedNormal = -frontFacedNormal;
    }

    vec3f w_t;
    const bool tir = !refract(w_t, triangleData.rayDir, frontFacedNormal, eta);
    const float cos_theta_t = -dot(frontFacedNormal, w_t);
    float R = tir ? 1.f : fresnel(cos_theta_i, cos_theta_t, eta);

    const float z = prd.random();
    if(z <= R)
    {
        //Reflect
        const vec3f w_in = reflect(normalize(triangleData.rayDir), normalize(frontFacedNormal));
        prd.nextRayDirection = w_in;
        my_strcat(prd.pathREGEX, "R");
    }
    else
    {
        //Refract
        const vec3f w_in = w_t;
        prd.nextRayDirection = w_in;
        my_strcat(prd.pathREGEX, "S");
    }
    prd.nextRayOrigin = triangleData.surfPos;

    prd.pixelColor *= transmittance;
    prd.depth++;
}

extern "C" __global__ void __closesthit__light()
{
    const TriangleMeshSBTData &sbtData
            = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();

    const TriangleData triangleData(sbtData);

#ifndef PARALLEL_LIGHT //area light
    prd.pixelColor *= sbtData.emissionColor;
    prd.depth++;
    prd.isEnd = true;
    my_strcat(prd.pathREGEX, "L");
#else //parallel light

    const vec3f lightSourceDir = -triangleData.Ns;  //the normal of the light plate

    if(dot(triangleData.rayDir, lightSourceDir) > 0.99f)  //not perfect parallel is ok
    {
        prd.pixelColor *= sbtData.emissionColor;
        prd.depth++;
        prd.isEnd = true;
        my_strcat(prd.pathREGEX, "L");
    }
    else //not parallel is considered miss
    {
        prd.isEnd = true;
        prd.depth++;
        prd.pixelColor *= missColor;
    }

#endif
}

extern "C" __global__ void __anyhit__light()
{
    const TriangleMeshSBTData &sbtData
            = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();
    PRD &prd = *getPRD<PRD>();

    const TriangleData triangleData(sbtData);

    const vec3f lightSourceDir = -triangleData.Ns;  //the normal of the light plate

#ifndef PARALLEL_LIGHT //area light
    prd.anyHitLight = true;
#else //parallel light
    if(dot(triangleData.rayDir, lightSourceDir) > 0.99f) prd.anyHitLight = true;
#endif
}

} // ::osc
